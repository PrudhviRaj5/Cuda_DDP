#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<time.h>
#include<iostream>
using namespace std;
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include""
#define pi 3.14159265359
double* Make2DDoubleArray(int arraySizeX, int arraySizeY) {
	double* theArray;
	theArray = (double*)calloc(arraySizeX*arraySizeY, sizeof(double));
	return theArray;
}
double* Makecudaarray(int arraySizeX, int arraySizeY){
	double* Array;
	hipMalloc((void**)&Array, arraySizeX*arraySizeY*sizeof(double));
	return Array;
}

__global__ void velmodify(double *d_X, double *d_Y, double *d_Z, double *d_U, double *d_V, int xno, int yno, double dtbyrhodh, double freq, double dt, int xsource, int ysource, int m, double *d_A, double *d_x, double *d_y){
	int i = blockIdx.x;
	int j = threadIdx.x;
	if (i <= xno - 1 && j <= yno)
		d_U[(i + 1)*(yno + 2) + j] += dtbyrhodh*(d_X[((i + 1)*(yno + 2)) + j + 1] - d_X[((i + 1)*(yno + 2)) + j] + d_Z[((i + 1)*(yno + 1)) + j] - d_Z[(i*(yno + 1)) + j]);
	if (i <= xno  && j <= yno - 1)
		d_V[(i*(yno + 2)) + j + 1] += dtbyrhodh*(d_Z[(i*(yno + 1)) + j + 1] - d_Z[(i*(yno + 1)) + j] + d_Y[((i + 1)*(yno + 2)) + j + 1] - d_Y[(i*(yno + 2)) + j + 1]);
	__syncthreads();
	if ((m*dt) <= (double)3 / freq){
		d_V[(ysource*(yno + 2)) + xsource] = (1 - cosf(2 * pi*freq*dt*m / 3))*cosf(2 * pi*freq*m*dt);

	}
	__syncthreads();
	if (i == 0 && j == 0){
		*d_x += (d_U[(85 * (yno + 2)) + 96] * dt);
		*d_y += (d_V[(85 * (yno + 2)) + 96] * dt);
		d_A[m - 1] = sqrt(d_x[0] * d_x[0] + d_y[0] * d_y[0]);
	}
}

__global__ void strmodify(double *d_X, double *d_Y, double *d_Z, double *d_U, double *d_V, int xno, int yno, double lambdaplus2mudtbydh, double lambdadtbydh, double dtmubydh){
	int i = blockIdx.x;
	int j = threadIdx.x;
	if (i <= xno - 1 && j <= yno - 1){
		d_X[((i + 1)*(yno + 2)) + j + 1] += lambdaplus2mudtbydh*(d_U[((i + 1)*(yno + 2)) + j + 1] - d_U[((i + 1)*(yno + 2)) + j]) + lambdadtbydh*(d_V[((i + 1)*(yno + 2)) + j + 1] - d_V[(i*(yno + 2)) + j + 1]);
		d_Y[((i + 1)*(yno + 2)) + j + 1] += lambdaplus2mudtbydh*(d_V[((i + 1)*(yno + 2)) + j + 1] - d_V[(i*(yno + 2)) + j + 1]) + lambdadtbydh*(d_U[((i + 1)*(yno + 2)) + j + 1] - d_U[((i + 1)*(yno + 2)) + j]);

		__syncthreads();

	}
	if (i <= xno && j <= yno)
		d_Z[(i*(yno + 1)) + j] += dtmubydh*(d_V[(i*(yno + 2)) + j + 1] - d_V[(i*(yno + 2)) + j] + d_U[((i + 1)*(yno + 2)) + j] - d_U[(i*(yno + 2)) + j]);
	__syncthreads();

}

int main(){
	clock_t start1, end1, start2, end2;
	double time_taken;
	start1 = clock();
	int p = 2667, lo = 6396, sh = 3103;
	double mu = (double)sh*sh*p, lambda = (double)lo*lo*p - 2 * mu, freq = 2.25*1e6, wavelength = (double)lo / freq, dh = (double)wavelength / 30.0, dt = (double)dh / (lo*1.5);
	int length = 20, breadth = 20, m;
	int xno = floor(length / (1000.0*dh)), yno = floor(breadth / (1000.0*dh));
	xno = xno + 16 - (xno % 16);
	yno = yno + 16 - (yno % 16);
	xno = xno - 2;
	yno = yno - 2;
	int xsource = ((xno + 2) / 2) - 1, ysource = 0;
	double time_total = pow(10.0, -6.0);
	double timesteps = ceil(10e-6 / dt);
	double dtbyrhodh = dt / (p*dh), lambdaplus2mudtbydh = (lambda + 2 * mu)*dt / dh, lambdadtbydh = lambda*dt / dh, dtmubydh = dt*mu / dh;
	double *X, *Y, *U, *V, *Z, *A, *x, *y;
	X = Make2DDoubleArray(xno + 2, yno + 2);
	Y = Make2DDoubleArray(xno + 2, yno + 2);
	Z = Make2DDoubleArray(xno + 1, yno + 1);
	U = Make2DDoubleArray(xno + 2, yno + 2);
	V = Make2DDoubleArray(xno + 2, yno + 2);
	A = Make2DDoubleArray(1, timesteps);
	x = Make2DDoubleArray(1, 1);
	y = Make2DDoubleArray(1, 1);
	double* d_X = Makecudaarray(xno + 2, yno + 2);
	double* d_Y = Makecudaarray(xno + 2, yno + 2);
	double* d_Z = Makecudaarray(xno + 1, yno + 1);
	double* d_U = Makecudaarray(xno + 2, yno + 2);
	double* d_V = Makecudaarray(xno + 2, yno + 2);
	double* d_A = Makecudaarray(1, timesteps);
	double* d_x = Makecudaarray(1, 1);
	double* d_y = Makecudaarray(1, 1);
	hipMemcpy(d_X, X, (xno + 2)*(yno + 2)* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, (xno + 2)*(yno + 2)* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, Z, (xno + 1)*(yno + 1)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_U, U, (xno + 2)*(yno + 2)* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, (xno + 2)*(yno + 2)* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_A, A, (timesteps)* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, sizeof(double), hipMemcpyHostToDevice);
	dim3 threadsperblock(yno + 2);
	dim3 numblocks(xno + 2);
	end1 = clock();
	start2 = clock();
	for (m = 1; m <= timesteps; m++){

		velmodify << <numblocks, threadsperblock >> >(d_X, d_Y, d_Z, d_U, d_V, xno, yno, dtbyrhodh, freq, dt, xsource, ysource, m, d_A, d_x, d_y);

		strmodify << <numblocks, threadsperblock >> >(d_X, d_Y, d_Z, d_U, d_V, xno, yno, lambdaplus2mudtbydh, lambdadtbydh, dtmubydh);


	}
	end2 = clock();
	hipDeviceSynchronize();
	hipMemcpy(A, d_A, (timesteps)* sizeof(double), hipMemcpyDeviceToHost);


	for (int i = 0; i<timesteps; i++)
		printf("%d\t%e\n", i + 1, A[i]);


	time_taken = (double)(end1 - start1) / CLOCKS_PER_SEC;
	printf("Time elapsed is %lfseconds\nGRID Size:%d*%d\nTime Steps Taken:%lf\nNo of blocks:%d\n", time_taken, xno + 2, yno + 2, timesteps, yno + 2);
	printf("Time elapsed per function in gpu is %lf secomds\n", (double)(end2 - start2) / CLOCKS_PER_SEC);
	free(X);
	free(Y);
	free(Z);
	free(U);
	free(V);
	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);
	hipFree(d_U);
	hipFree(d_V);
	hipFree(d_A);
	hipFree(d_x);
	hipFree(d_y);
	getchar();
	return 0;
}
